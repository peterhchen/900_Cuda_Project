
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda()
{
    printf("Hello Cuda world\n");
}

int main()
{
    // case 1:
    /*hello_cuda << < 1,1 > >> ();*/
    /*hello_cuda << < 1,2 > >> ();*/
    // case 2:
    // 3 x 2
    //dim3 block(2, 1, 1);
    //dim3 grid(3, 1, 1);
    
    //case 3:
    // 2x2, 8x2
    // dim3 block(8, 2, 1);
    // dim3 grid(2, 2, 1);

    int nx = 16, ny = 4;
    dim3 block(8, 2, 1);
    dim3 grid(nx / block.x, ny / block.y);
    hello_cuda <<< grid, block >>> ();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}