
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_details ()
{
    printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx: %d, \
        blockDim.x: %d, blockDim.y: %d, blockDim: %d, \
        gridDim.x: %d, gridDim.y: %d, girdDim: %d\n", \
        blockIdx.x, blockIdx.y, blockIdx.z, \
        blockDim.x, blockDim.y, blockDim.z, \
        gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
    int nx = 16, ny = 16;
    dim3 block(8, 8);
    dim3 grid(nx / block.x, ny / block.y);
    print_details <<< grid, block >>> ();
    hipDeviceSynchronize();
    int cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    hipDeviceReset();
    return 0;
}
