#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
__global__ void unique_idx_calc_threadIdx(int *input)
{
    int tid = threadIdx.x;
    printf("threadIdx: %d, value: %d\n", tid, input[tid]);
}
__global__ void unique_gid_calculation(int* input)
{
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;
    printf("blockIdx: %d, threadIdx.x: %d, gid: %d, value: %d\n", 
        blockIdx, tid, gid, input[gid]);
}
int main()
{
    /* case 1 and 2:
    int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33 };*/
    int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 12, 342, 56, 44, 99};
    int array_size = sizeof (h_data)/sizeof(int);
    printf("array_size: %d\n\n", array_size);
    int array_byte_size = sizeof(int) * array_size;
    for (int i = 0; i < array_size; i++) {
        printf("%d ", h_data[i]);
    }
    printf("\n\n");
    int * d_data;
    hipMalloc((void **)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
    /*  case 1: one block in grid and 8 threads in each block
    dim3 block(8);
    dim3 grid(1);*/
    // case 2: two blocks in grid and 4 threds in each block
    /*dim3 block(4);
    dim3 grid(2);
    unique_idx_calc_threadIdx << < grid, block >> > (d_data);
    */
    // case 3: 4 block in grid and 4 thread in each block
    dim3 block(4);
    dim3 grid(4);
    unique_gid_calculation << < grid, block >> > (d_data);
    hipDeviceSynchronize();
    int cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}
