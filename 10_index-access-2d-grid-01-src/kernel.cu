#include "hip/hip_runtime.h"
// Case 2: 2D index
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void unique_gid_calculation_2d(int* data)
{
    int tid = threadIdx.x;
    int block_offset = blockIdx.x * blockDim.x;
    int row_offset = blockDim.x * gridDim.x * blockIdx.y;
    int gid = row_offset + block_offset + tid;
    printf("Case 2: blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, gid: %d, value: %d\n",
        blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}
int main()
{
    int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 22, 43, 56, 4, 76, 81, 94, 32};
    int array_size = sizeof(h_data) / sizeof(int);
    printf("case 2: array_size: %d\n\n", array_size);
    int array_byte_size = sizeof(int) * array_size;
    for (int i = 0; i < array_size; i++) {
        printf("%d ", h_data[i]);
    }
    printf("\n\n");
    int* d_data;
    hipMalloc((void**)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
    // we have 4 threads in each block.
    // we have 2x2 blocks in grid.
    dim3 block(4);
    dim3 grid(2, 2);
    unique_gid_calculation_2d << < grid, block >> > (d_data);
    hipDeviceSynchronize();
    int cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}


// case 1:
//#include "hip/hip_runtime.h"
//#include ""
//#include <stdio.h>
//
//__global__ void unique_gid_calculation_2d(int* data)
//{
//    int tid = threadIdx.x;
//    int offset = blockIdx.x * blockDim.x;
//    int gid = tid + offset;
//    printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, gid: %d, value: %d\n",
//        blockIdx.x, blockIdx.y, tid, gid, data[gid]);
//}
//int main()
//{
//    int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 22, 43, 56, 4, 76, 81, 94, 32};
//    int array_size = sizeof(h_data) / sizeof(int);
//    printf("array_size: %d\n\n", array_size);
//    int array_byte_size = sizeof(int) * array_size;
//    for (int i = 0; i < array_size; i++) {
//        printf("%d ", h_data[i]);
//    }
//    printf("\n\n");
//    int* d_data;
//    hipMalloc((void**)&d_data, array_byte_size);
//    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
//    // we have 4 threads in each block.
//    // we have 2x2 blocks in grid.
//    dim3 block(4);
//    dim3 grid(2, 2);
//    unique_gid_calculation_2d << < grid, block >> > (d_data);
//    hipDeviceSynchronize();
//    int cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//    return 0;
//}
